#include "hip/hip_runtime.h"
/** Survey propagation -*- C++ -*-
 * @file
 * @section License
 *
 * Galois, a framework to exploit amorphous data-parallelism in irregular
 * programs.
 *
 * Copyright (C) 2013, The University of Texas at Austin. All rights reserved.
 * UNIVERSITY EXPRESSLY DISCLAIMS ANY AND ALL WARRANTIES CONCERNING THIS
 * SOFTWARE AND DOCUMENTATION, INCLUDING ANY WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR ANY PARTICULAR PURPOSE, NON-INFRINGEMENT AND WARRANTIES OF
 * PERFORMANCE, AND ANY WARRANTY THAT MIGHT OTHERWISE ARISE FROM COURSE OF
 * DEALING OR USAGE OF TRADE.  NO WARRANTY IS EITHER EXPRESS OR IMPLIED WITH
 * RESPECT TO THE USE OF THE SOFTWARE OR DOCUMENTATION. Under no circumstances
 * shall University be liable for incidental, special, indirect, direct or
 * consequential damages or loss of profits, interruption of business, or
 * related expenses which may arise from use of Software or Documentation,
 * including but not limited to those resulting from defects in Software and/or
 * Documentation, or loss or inaccuracy of data of any kind.
 *
 * @section Description
 *
 * Implementation of the Survey Propagation Algorithm
 *
 * @author Sreepathi Pai <sreepai@ices.utexas.edu>
 */

/* -*- mode: C++ -*- */
#include "common.h"

#include "lonestargpu/lonestargpu.h"
#include "lonestargpu/cuda_launch_config.hpp"
#ifndef _KLAP_
#include "lonestargpu/cutil_subset.h"
#include <cub/hipcub/hipcub.hpp>
#else
#define CUDA_SAFE_CALL(x)

namespace cub {
    template<typename T, int i>
    class BlockReduce {
        public:
        typedef int TempStorage;
        BlockReduce(TempStorage t);
        float Sum(float);
    };
    template<typename T>
    class DoubleBuffer {
        public:
        DoubleBuffer(T*,T*);
        T* Current();
    };
    class DeviceRadixSort {
        public:
        static void SortPairs(void*, size_t, hipcub::DoubleBuffer<float>&, hipcub::DoubleBuffer<int>, int);
    };
}
#endif

#define WARP_SIZE 32
//__device__ inline int lane_id(void) { return threadIdx.x % WARP_SIZE; }

#define EPSILON 0.01 /* in the source, in the paper 10^-3 */
#define MAXITERATION 1000
#define PARTIAL "outputs/partial.cnf"
#define PARAMAGNET 0.01

KernelConfig kc;
const int nSM = kc.getNumberOfSMs();

void init_from_file(const char *F,
        int max_lit_per_clause,
        struct CSRGraph &clauses,
        struct CSRGraph &vars,
        struct Edge &ed)
{
    FILE *f = fopen(F, "r");

    int nclauses, nvars, ret;
    char line[255];

    if(!f)
    {
        fprintf(stderr, "unable to read file %s.\n", F);
        exit(1);
    }

    while(true)
    {
        if(fgets(line, 255, f))
        {
            if(line[0] != 'c')
                break;
            printf("%s", line);
        }
        else
        {
            fprintf(stderr, "unable to read %s\n", F);
            exit(1);
        }
    }

    ret = sscanf(line, "p cnf %d %d", &nvars, &nclauses);
    assert(ret == 2);

    clauses.nnodes = nclauses;
    vars.nnodes = nvars;
    ed.nedges = clauses.nedges = vars.nedges = nclauses * max_lit_per_clause; // over-estimate

    assert(clauses.alloc());
    assert(vars.alloc());
    assert(ed.alloc());

    int newlit, lit;
    int clndx = 0, litndx = 0, edndx = 0;

    /* read lines of literals terminated by 0 */
    /* assumes literals numbered from 1 */

    do {
        ret = fscanf(f, "%d", &newlit);
        if(ret == EOF) break;

        if(newlit == 0)
        {
            assert(clndx < nclauses);

            clndx++;
            litndx = 0;

            clauses.row_offsets[clndx] = edndx;
            continue;
        }

        assert(litndx < max_lit_per_clause);

        // convert to zero-based
        lit = ((newlit < 0) ? -newlit : newlit) - 1;

        assert(lit >= 0);

        ed.src[edndx] = clndx;
        ed.dst[edndx] = lit;
        ed.bar[edndx] = newlit < 0;
        ed.eta[edndx] = (float)(rand()) / (float)RAND_MAX;

        // essentially clause -> edge
        clauses.columns[clauses.row_offsets[clndx] + litndx] = edndx;

        // record size of every var node
        vars.row_offsets[lit]++;

        litndx++;
        edndx++;
    } while(true);
    fclose(f);

    clauses.nedges = vars.nedges = ed.nedges = edndx;

    clauses.set_last_offset();
    vars.set_last_offset();

    /* populate vars */
    // exclusive-sum
    for(int i = 0, sum = 0; i < vars.nnodes; i++)
    {
        int size = vars.row_offsets[i];
        vars.row_offsets[i] = sum;
        sum += size;
    }

    int *varndx = (int *) calloc(vars.nedges, sizeof(int));

    for(int i = 0; i < ed.nedges; i++)
    {
        unsigned var = ed.dst[i];
        vars.columns[vars.row_offsets[var] + varndx[var]++] = i;
    }

    //printf("read %d clauses, %d variables, %d literals\n", clauses.nnodes, vars.nnodes, ed.nedges);
}

void print_solution(const char *sol, const CSRGraph &vars)
{
    FILE *f = fopen(sol, "w");
    int i;
    for(i = 0; i < vars.nnodes; i++)
    {
        if(vars.sat[i])
            fprintf(f, "%d\n", vars.value[i] ? (i + 1) : -(i + 1));
    }
    fclose(f);
}

void dump_formula(const char *output, const CSRGraph &clauses, const CSRGraph &vars, const Edge &ed)
{
    FILE *of = fopen(output, "w");

    fprintf(of, "p cnf %d %d\n", vars.nnodes, clauses.nnodes);

    for(int cl = 0; cl < clauses.nnodes; cl++) {
        unsigned offset = clauses.row_offsets[cl];

        for(int i = 0; i < clauses.degree(cl); i++) {
            unsigned edndx = clauses.columns[offset + i];
            fprintf(of, "%d ", ed.bar[edndx] ? -(ed.dst[edndx]+1) : (ed.dst[edndx]+1));
        }
        fprintf(of, "0\n");
    }
}

void dump_partial(const char *output, const CSRGraph &clauses, const CSRGraph &vars, const Edge &ed)
{
    FILE *of = fopen(output, "w");

    int sat = 0;
    for(int cl = 0; cl < clauses.nnodes; cl++)
        if(clauses.sat[cl]) sat++;

    fprintf(of, "p cnf %d %d\n", vars.nnodes, clauses.nnodes - sat);

    for(int cl = 0; cl < clauses.nnodes; cl++) {
        if(clauses.sat[cl])
            continue;

        unsigned offset = clauses.row_offsets[cl];

        for(int i = 0; i < clauses.degree(cl); i++) {
            unsigned edndx = clauses.columns[offset + i];

            if(vars.sat[ed.dst[edndx]])
                continue;

            fprintf(of, "%d ", ed.bar[edndx] ? -(ed.dst[edndx]+1) : (ed.dst[edndx]+1));
        }
        fprintf(of, "0\n");
    }
}

__global__ void calc_pi_values(GPUCSRGraph clauses, GPUCSRGraph vars, Edge ed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int threads = blockDim.x * gridDim.x;

    // over all a -> j
    for(int edndx = id; edndx < ed.nedges; edndx += threads)
    {
        int j = ed.dst[edndx];
        int a = ed.src[edndx];

        if(clauses.sat[a] || vars.sat[j])
            continue;

        int V_j = vars.row_offsets[j];
        int V_j_len = vars.degree(j);

        float pi_0 = 1.0;
        float V_s_a = 1.0;
        float V_u_a = 1.0;

        // over all b E V(j)
        for(int i = 0; i < V_j_len; i++)
        {
            int ed_btoj = vars.columns[V_j + i];

            int b = ed.src[ed_btoj];

            if(clauses.sat[b])
                continue;

            if(b != a)
            {
                pi_0 *= (1 - ed.eta[ed_btoj]);

                if(ed.bar[ed_btoj] == ed.bar[edndx])
                    V_s_a *= (1 - ed.eta[ed_btoj]);
                else
                    V_u_a *= (1 - ed.eta[ed_btoj]);
            }
        }

        ed.pi_0[edndx] = pi_0;
        ed.pi_U[edndx] = (1 - V_u_a) * (V_s_a);
        ed.pi_S[edndx] = (1 - V_s_a) * (V_u_a);

        //printf("%f %f %f\n", ed.pi_0[edndx,
    }
}

__global__ void update_eta(GPUCSRGraph clauses, GPUCSRGraph vars, Edge ed, float *max_eps)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int threads = blockDim.x * gridDim.x;
    float eps, lmaxeps = 0;

    for(int edndx = id; edndx < ed.nedges; edndx+=threads)
    {
        int a = ed.src[edndx];
        int i = ed.dst[edndx];

        // as these are "removed"
        if(clauses.sat[a] || vars.sat[i])
            continue;

        int clndx = clauses.row_offsets[a];
        int nlit = clauses.degree(a);

        float new_eta = 1.0;

        for(int aedndx = 0; aedndx < nlit; aedndx++)
        {
            int jedndx = clauses.columns[clndx + aedndx];

            int j = ed.dst[jedndx];

            if(j == i)
                continue;

            if(vars.sat[j])
                continue;

            float sum = ed.pi_0[jedndx] + ed.pi_S[jedndx] + ed.pi_U[jedndx];

            if(sum == 0.0) { // TODO: non-standard ...
                new_eta = 0;
                break;
            }

            new_eta *= ed.pi_U[jedndx] / sum;
        }

        eps = fabs(new_eta - ed.eta[edndx]);
        if(eps > lmaxeps)
            lmaxeps = eps;

        ed.eta[edndx] = new_eta;
    }

    if(lmaxeps)
        atomicMax((int *) max_eps, __float_as_int(lmaxeps));
}

__global__ void update_bias(GPUCSRGraph clauses, GPUCSRGraph vars, Edge ed, float *bias_list, int *bias_list_vars, int *bias_len, float *g_summag)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int threads = blockDim.x * gridDim.x;
    float maxmag;
    float summag = 0;

    for(int v = id; v < vars.nnodes; v+=threads)
    {
        if(vars.sat[v])
            continue;

        float pi_0_hat = 1.0, V_minus = 1.0, V_plus = 1.0;
        float pi_P_hat, pi_M_hat;

        int edoff = vars.row_offsets[v];
        int ncl = vars.degree(v);

        // a E v(i)
        for(int edndx = 0; edndx < ncl; edndx++)
        {
            int edge = vars.columns[edoff + edndx];
            int cl = ed.src[edge];

            if(clauses.sat[cl])
                continue;

            pi_0_hat *= (1 - ed.eta[edge]);

            if(ed.bar[edge])
                V_minus *= (1 - ed.eta[edge]);
            else
                V_plus *= (1 - ed.eta[edge]);
        }

        pi_P_hat = (1 - V_plus) * V_minus;
        pi_M_hat = (1 - V_minus) * V_plus;

        float W_plus, W_minus; //, W_zero;

        if (((pi_0_hat + pi_P_hat + pi_M_hat)) == 0.0)
        {
            W_plus = 0.0;
            W_minus = 0.0;
        }
        else
        {
            W_plus = pi_P_hat / (pi_0_hat + pi_P_hat + pi_M_hat);
            W_minus = pi_M_hat / (pi_0_hat + pi_P_hat + pi_M_hat);
        }

        //W_zero = 1 - W_plus - W_minus;

        vars.bias[v] = fabs(W_plus - W_minus);
        vars.value[v] = (W_plus > W_minus);

        maxmag = W_plus > W_minus ? W_plus : W_minus;
        summag += maxmag;

        int ndx = atomicAdd(bias_len, 1);
        bias_list[ndx] = vars.bias[v];
        bias_list_vars[ndx] = v;
    }

    typedef hipcub::BlockReduce<float, 384> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    summag = BlockReduce(temp_storage).Sum(summag);

    if(threadIdx.x == 0)
        atomicAdd(g_summag, summag);
}

int compare_float(const void *x, const void *y)
{
    float xx = *(float *)x, yy = *(float *)y;

    // reverse order
    if(xx > yy)
        return -1;
    else if (xx < yy)
        return 1;

    return 0;
}

float sort_bias_list(hipcub::DoubleBuffer<float> &db_bias_list,
        hipcub::DoubleBuffer<int> &db_bias_list_vars,
        int *g_bias_list_len, float summag, int& fixperstep)
{
    int bias_list_len;
    static void *d_temp_storage = NULL;
    static size_t temp_storage_bytes = 0;

    CUDA_SAFE_CALL(hipMemcpy(&bias_list_len, g_bias_list_len, 1 * sizeof(int), hipMemcpyDeviceToHost));

    float r = 0;

    if(bias_list_len)
    {
        r = (summag / bias_list_len);

#if PRINT
        printf("<bias>:%f\n", r);
#endif

        if(r < PARAMAGNET)
            return r;

        if(d_temp_storage == NULL)
        {
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, db_bias_list,
                    db_bias_list_vars, bias_list_len);

            // Allocate temporary storage for sorting operation
            CUDA_SAFE_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
        }

        // Run sorting operation
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, db_bias_list,
                db_bias_list_vars, bias_list_len);

        if(fixperstep > bias_list_len)
            fixperstep = 1;
    }

    return r;
}

int converge(GPUCSRGraph &g_cl, GPUCSRGraph &g_vars, Edge &g_ed, float *g_max_eps)
{
    float max_eps;
    int i = 0;
    const size_t cpv_res = maximum_residency(calc_pi_values, 384, 0);//Comment = 10
    const size_t ue_res = maximum_residency(update_eta, 384, 0);//Comment = 10...

    do {
        calc_pi_values<<<nSM * cpv_res, 384>>>(g_cl, g_vars, g_ed);

        max_eps = 0;
        CUDA_SAFE_CALL(hipMemcpy(g_max_eps, &max_eps,
                    sizeof(float), hipMemcpyHostToDevice));

        update_eta<<<nSM * ue_res, 384>>>(g_cl, g_vars, g_ed, g_max_eps);

        CUDA_SAFE_CALL(hipMemcpy(&max_eps, g_max_eps,
                    sizeof(float), hipMemcpyDeviceToHost));

    } while(max_eps > EPSILON && i++ < MAXITERATION);

    if(max_eps <= EPSILON) {
#if PRINT
        printf("converged in %d iterations max eps %f\n", i, max_eps);
#endif
        return 1;
    } else {
        printf("SP UN-CONVERGED, max eps %f\n", max_eps);
        //TODO write out formula?
        exit(1);
    }

    return 0;
}

int build_list(GPUCSRGraph &g_cl, GPUCSRGraph &g_vars, Edge &g_ed, float *g_summag,
        hipcub::DoubleBuffer<float> &db_bias_list, hipcub::DoubleBuffer<int> &db_bias_list_vars,
        int *g_bias_list_len, int &fixperstep)
{
    float summag;
    int bias_list_len;
    static size_t updb_res = maximum_residency(update_bias, 384, 0);//Comment

    summag = 0;
    CUDA_SAFE_CALL(hipMemcpy(g_summag, &summag, sizeof(summag), hipMemcpyHostToDevice));

    bias_list_len = 0;
    CUDA_SAFE_CALL(hipMemcpy(g_bias_list_len, &bias_list_len, sizeof(int) * 1, hipMemcpyHostToDevice));
    update_bias<<<nSM * updb_res, 384>>>(g_cl, g_vars, g_ed, db_bias_list.Current(),
            db_bias_list_vars.Current(), g_bias_list_len, g_summag);
    CUDA_SAFE_CALL(hipMemcpy(&summag, g_summag, sizeof(summag), hipMemcpyDeviceToHost));

    float limitbias = sort_bias_list(db_bias_list, db_bias_list_vars, g_bias_list_len, summag, fixperstep);
    if(limitbias < PARAMAGNET)
    {
        //printf("paramagnetic state\n");
        return 1;
    }

    return 0;
}

int compare_solutions(const CSRGraph &vars, const CSRGraph &vars_, char* version)
{
    int error = 0;
    int i;
    for(i = 0; i < vars.nnodes; i++)
    {
        if(vars.sat[i]){
            //printf("Version %s vars %d -> %d - %d\n", version, i, vars.value[i], vars_.value[i]);
            if(vars.value[i] != vars_.value[i]){
                error = 1;
                printf("Error in version %s vars %d -> %d != %d\n", version, i, vars.value[i], vars_.value[i]);
            }
        }
    }
    return error;
}

//********************************************************************************************************************


int main(int argc, char *argv[])
{
    // Parameters
    int warmup              = 1;
    int runs                = 3;
    int outputLevel         = 1;
    const char* fileName    = "inputs/random-42000-10000-3.cnf";
    int max_literals        = 10000;
    int opt;
    while((opt = getopt(argc, argv, "w:r:o:f:m:h")) >= 0) {
        switch(opt) {
            case 'w': warmup        = atoi(optarg); break;
            case 'r': runs          = atoi(optarg); break;
            case 'o': outputLevel   = atoi(optarg); break;
            case 'f': fileName      = optarg;       break;
            case 'm': max_literals  = atoi(optarg); break;
            default : std::cerr <<
                      "\nUsage:  ./bt [options]"
                          "\n"
                          "\n    -w <W>    # of warmup runs (default=1)"
                          "\n    -r <R>    # of timed runs (default=3)"
                          "\n    -o <O>    level of output verbosity (0: one CSV row, 1: moderate, 2: verbose)"
                          "\n    -f <F>    file name (default=inputs/random-42000-10000-3.cnf)"
                          "\n    -m <M>    maximum literals (default=10000)"
                          "\n    -h        help\n\n";
                      exit(0);
        }
    }

    //srand(7);
    double starttime, endtime, runtime;
    CSRGraph cl, vars;
    Edge ed;
    GPUCSRGraph g_cl, g_vars;
    GPUEdge g_ed;
    float *g_max_eps;
    float *g_bias_list, *g_bias_list_2;
    int *g_bias_list_vars, *g_bias_list_vars_2;
    int *g_bias_list_len;
    float *g_summag;
    const size_t d2_res = maximum_residency(decimate, 384, 0);
    init_from_file(fileName, max_literals, cl, vars, ed);
    g_vars.from_cpu(vars);
    CUDA_SAFE_CALL(hipMalloc(&g_max_eps, sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&g_summag, sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&g_bias_list, g_vars.nnodes * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&g_bias_list_2, g_vars.nnodes * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&g_bias_list_vars, g_vars.nnodes * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc(&g_bias_list_vars_2, g_vars.nnodes * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc(&g_bias_list_len, sizeof(int)));
    hipcub::DoubleBuffer<float> db_bias_list(g_bias_list, g_bias_list_2);
    hipcub::DoubleBuffer<int> db_bias_list_vars(g_bias_list_vars, g_bias_list_vars_2);
    hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, d2_res*nSM*384); // Fixed-size pool

    float totalKernelTime = 0;
    for(int run = -warmup; run < runs; run++) {

        if(outputLevel >= 1) {
            if(run < 0) {
                std::cout << "Warmup:\t";
            } else {
                std::cout << "Run " << run << ":\t";
            }
        }

        g_cl.from_cpu(cl);
        g_ed.from_cpu(ed);
        int canfix = 0.01 * vars.nnodes;
        if(canfix < 1) canfix = 1;
        int round;
        int breaker = 54;
        //int error = 0;
        runtime = 0;
        round = 0;
        // Start execution
        while(converge(g_cl, g_vars, g_ed, g_max_eps)) {
            //printf("round = %d\n", round++);
            if(round == breaker) break;

            if(build_list(g_cl, g_vars, g_ed, g_summag,
                        db_bias_list, db_bias_list_vars,
                        g_bias_list_len,
                        canfix))
                break;

            CUDA_SAFE_CALL(hipDeviceSynchronize());
            starttime = rtclock();

            launch_kernel(d2_res * nSM, 384, g_cl, g_vars, g_ed,
                    db_bias_list_vars.Current(), g_bias_list_len, canfix);

            CUDA_SAFE_CALL(hipDeviceSynchronize());
            endtime = rtclock();
            runtime += (1000.0 * (endtime - starttime));
        };

        if(outputLevel >= 1) std::cout << "run kernel time = " << runtime << std::endl;
        if (run >= 0) totalKernelTime += runtime;
        g_cl.to_cpu(cl);
        g_vars.to_cpu(vars);
        g_ed.to_cpu(ed);
        print_solution("outputs/sp_sol.dat", vars);
        //char version[10];
        //strcpy(version, "CDP");
        //error = compare_solutions(vars_gold, vars, version);
        //if(error) printf("Test failed %s\n", version);
        //else printf("Test passed %s\n", version);
        dump_partial(PARTIAL, cl, vars, ed);

        if(run < runs) {
            init_from_file(fileName, max_literals, cl, vars, ed);
            g_vars.from_cpu(vars);
        }

    }

    if(outputLevel >= 1) {
        std::cout<< "Average kernel time = " << totalKernelTime/runs << " ms\n";
    } else {
        std::cout<< totalKernelTime/runs;
    }

}

